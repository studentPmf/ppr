#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__global__ void funkc(int *M, int dim, unsigned int *fsum)
{
  unsigned int rez;
  extern __shared__ int sum[];

  sum[blockIdx.x*gridDim.x + blockIdx.y] = 0;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[dim+i+j]*M[dim*i+j];
  else
    rez = 0;

   atomicAdd(&sum[blockIdx.x*gridDim.x + blockIdx.y], rez);
   //__syncthreads();

   fsum[blockIdx.x*gridDim.x + blockIdx.y] =  sum[blockIdx.x*gridDim.x + blockIdx.y] ;
   //__syncthreads();
}

__global__ void vecAdd(unsigned int *M, int dim, int *r)
{
  __shared__ int rez;
  rez = 0;
  __syncthreads();
  int i = threadIdx.x;
  int value(0);
  if(i < dim)
    value = M[i];
  
  atomicAdd(&rez, value);
  
  __syncthreads();
  r[0] = rez;
}

int main(int argc, char*argv[])
{
  int N(100);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  for(int i(0); i < N*N; i++)
    M_h[i] = 1;//i%3; // elements in the matrix is less than 3

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(32,32);
  dim3 blocksPerGrid((N/threadsPerBlock.x) + 1, (N/threadsPerBlock.y) + 1);
  int gridDimension = blocksPerGrid.x*blocksPerGrid.y;
  cout<<blocksPerGrid.x<<","<<blocksPerGrid.y<<endl;
  int *result = (int*)malloc(gridDimension*sizeof(int));
  unsigned int *fsum;
  hipMalloc(&fsum, gridDimension*sizeof(int));
  funkc<<<blocksPerGrid, threadsPerBlock, gridDimension>>>(M_d, N, fsum);
  hipMemcpy(result, fsum, gridDimension*sizeof(int), hipMemcpyDeviceToHost);
  int *r;
  hipMalloc(&r, sizeof(int));
  vecAdd<<<1, gridDimension>>>(fsum, gridDimension, r);
  int rezz = (int*)malloc(sizeod(int));
  hipMemcpy(rezz, r, sizeof(int), hipMemcpyDeviceToHost);

  cout<<endl<<"presuma = "<<rezz[0]<<endl;
  int suma = 0;
  for (int s(0); s < gridDimension; s++)
  {
    suma+=result[s];
    cout<<"rezultat je:"<<result[s]<<endl;
  }
  
  cout<<endl<<"Konacan rezultat je : "<<sqrt(suma)<<endl;

  free(M_h);
  hipFree(M_d);
  hipFree(fsum);

  return 0;
}
