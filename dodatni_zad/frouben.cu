#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

/*
  suma elemenata po blokovima koristenjem aomic funkcije
*/
__global__ void funkc(int *M, int dim, unsigned int *fsum)
{
  unsigned int rez;
  extern __shared__ int sum[];

  sum[blockIdx.x*gridDim.x + blockIdx.y] = 0;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[dim+i+j]*M[dim*i+j];
  else
    rez = 0;

   atomicAdd(&sum[blockIdx.x*gridDim.x + blockIdx.y], rez);
   //__syncthreads();

   fsum[blockIdx.x*gridDim.x + blockIdx.y] =  sum[blockIdx.x*gridDim.x + blockIdx.y] ;
   //__syncthreads();
}

/*
   suma elemenata vektora dobivenog kao suma kvadrata elemenata blokova prethodne funkcije
*/
__global__ void vecAdd(unsigned int *M, int dim, int *r)
{
  __shared__ int rez;
  rez = 0;
  __syncthreads();
  
  int i = threadIdx.x;
  int value(0);
  
  if(i < dim)
    value = M[i];
  
  atomicAdd(&rez, value);
  
  __syncthreads();
  r[0] = rez;
}



int main(int argc, char*argv[])
{
  int N(100);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  // geneiranje matrice
  for(int i(0); i < N*N; i++)
    M_h[i] = i%3; // elements in the matrix is less than 3

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  // postavljanje topologije 32x32 threada u bloku i onda blokova koliko treba
  dim3 threadsPerBlock(32,32);
  dim3 blocksPerGrid((N/threadsPerBlock.x) + 1, (N/threadsPerBlock.y) + 1);

  int gridDimension = blocksPerGrid.x*blocksPerGrid.y;

  int *result = (int*)malloc(gridDimension*sizeof(int));
  unsigned int *fsum;
  hipMalloc(&fsum, gridDimension*sizeof(int));

  // poziv funkcije za racunanje sume kvadrata po blokovima
  funkc<<<blocksPerGrid, threadsPerBlock, gridDimension>>>(M_d, N, fsum);

  hipMemcpy(result, fsum, gridDimension*sizeof(int), hipMemcpyDeviceToHost);
  int *vectorSum;
  hipMalloc(&vectorSum, sizeof(int));
  
  // poziv funkcije za racunanje zbroja dobivenih suma
  vecAdd<<<1, gridDimension>>>(fsum, gridDimension, vectorSum);
  int *sumAll = (int*)malloc(sizeof(int));
  hipMemcpy(sumAll, vectorSum, sizeof(int), hipMemcpyDeviceToHost);

  /*for (int s(0); s < gridDimension; s++)
  {
    cout<<"rezultat je:"<<result[s]<<endl;
  }*/

  cout<<endl<<"Konacan rezultat je : "<<sqrt(sumAll[0])<<endl;

  free(M_h);
  free(sumAll);
  hipFree(M_d);
  hipFree(fsum);
  hipFree(r);

  return 0;
}
