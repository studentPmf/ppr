#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__global__ void funkc(int *M, int dim, unsigned int *fsum)
{
  unsigned int rez;
  __shared__ unsigned int sum;

  sum = 0;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[dim+i+j]*M[dim*i+j];
  else
    rez = 0;

   atomicAdd((int*)&sum, rez);
   //__syncthreads();

   fsum[blockIdx.x*blockDim.x + blockIdx.y] = sum;
   //__syncthreads();
}


int main(int argc, char*argv[])
{
  int N(50);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  for(int i(0); i < N*N; i++)
    M_h[i] = 1;//i%3; // elements in the matrix is less than 3

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(32,32);
  dim3 blocksPerGrid((N/threadsPerBlock.x) + 1, (N/threadsPerBlock.y) + 1);
  int gridDimension = blocksPerGrid.x*blocksPerGrid.y;
  cout<<blocksPerGrid.x<<","<<blocksPerGrid.y<<endl;
  int *result = (int*)malloc(gridDimension*sizeof(int));
  unsigned int *fsum;
  hipMalloc(&fsum, gridDimension*sizeof(int));
  funkc<<<blocksPerGrid, threadsPerBlock>>>(M_d, N,fsum);
  hipMemcpy(result, fsum, 1*sizeof(int), hipMemcpyDeviceToHost);
  for (int s(0); s < gridDimension; s++)
    cout<<"rezultat je:"<<result[s]<<endl;

  free(M_h);
  hipFree(M_d);
  hipFree(fsum);

  return 0;
}
