#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__global__ void funkc(int *M, int dim, unsigned int *fsum)
{
  unsigned int rez;
  __shared__ unsigned int sum = 0;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[dim+i+j]*M[dim*i+j];

   __syncthreads();
   atomicAdd(&sum, rez);
   __syncthreads();

   fsum[0] = sum;
}


int main(int argc, char*argv[])
{
  int N(100);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  for(int i(0); i < N*N; i++)
    M_h[i] = i%3;

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16,16);
  int blocksPerGrid = ((N / threadsPerBlock.x) + 1, (N/ threadsPerBlock.y) + 1);
  int result;
  int *fsum;
  hipMalloc(&fsum, 1*sizeof(int));
  funkc<<<blocksPerGrid, threadsPerBlock>>>(M_d, N,fsum);
  hipMemcpy(&result, fsum, 1*sizeof(int), hipMemcpyDeviceToHost);
  cout<<"rezultat je:"<<result<<endl;

  free(M_h);
  hipFree(M_d);
  hipFree(fsum);

  return 0;
}
