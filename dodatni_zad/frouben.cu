#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__global__ void funkc(int *M, int dim, unsigned int *fsum)
{
  unsigned int rez;
  extern __shared__ int sum[];

  //sum[blockIdx.x*gridDim.x + blockIdx.y] = 0;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[dim+i+j]*M[dim*i+j];
  else
    rez = 0;

   atomicAdd(&sum[blockIdx.x*gridDim.x + blockIdx.y], rez);
   //__syncthreads();

   fsum[blockIdx.x*gridDim.x + blockIdx.y] = sum[blockIdx.x*gridDim.x + blockIdx.y] ;
   //__syncthreads();
}


int main(int argc, char*argv[])
{
  int N(100);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  for(int i(0); i < N*N; i++)
    M_h[i] = 1;//i%3; // elements in the matrix is less than 3

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16,16);
  dim3 blocksPerGrid((N/threadsPerBlock.x) + 1, (N/threadsPerBlock.y) + 1);
  int gridDimension = blocksPerGrid.x*blocksPerGrid.y;
  cout<<blocksPerGrid.x<<","<<blocksPerGrid.y<<endl;
  int *result = (int*)malloc(gridDimension*sizeof(int));
  unsigned int *fsum;
  hipMalloc(&fsum, gridDimension*sizeof(int));
  funkc<<<blocksPerGrid, threadsPerBlock,gridDimension>>>(M_d, N, fsum);
  hipMemcpy(result, fsum, 1*sizeof(int), hipMemcpyDeviceToHost);
  for (int s(0); s < gridDimension; s++)
    cout<<"rezultat je:"<<result[s]<<endl;

  free(M_h);
  hipFree(M_d);
  hipFree(fsum);

  return 0;
}
