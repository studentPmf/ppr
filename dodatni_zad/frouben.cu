#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__device__ unsigned int *fsum;

__global__ void funkc(int *M, int dim)
{
  unsigned int rez;
  __shared__ unsigned int sum;

  sum = 0;
  __syncthreads();

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[dim+i+j]*M[dim*i+j];
  else
    rez = 0;

   atomicAdd((int*)&sum, rez);
   __syncthreads();
   fsum[blockIdx.x*blockDim.x + blockIdx.y] = sum;
}

__global__ void VecAdd(unsigned int *rez, int dim)
{
  int i = threadIdx.x;
  int val(0);
  __shared__ unsigned int sum;
  sum = 0;
  __syncthreads();

  if(i < dim)
     val = fsum[i];

  atomicAdd((int*)&sum, val);
  __syncthreads();
  rez[0] = sum;
}

int main(int argc, char*argv[])
{
  int N(50);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  for(int i(0); i < N*N; i++)
    M_h[i] = 1;//i%3; // elements in the matrix is less than 3

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(32,32);
  dim3 blocksPerGrid((N/threadsPerBlock.x) + 1, (N/threadsPerBlock.y) + 1);
  cout<<blocksPerGrid.x<<","<<blocksPerGrid.y<<endl;
  int *result = (int*)malloc(sizeof(int));
  unsigned int *ptr;
  hipMalloc(&ptr, blocksPerGrid.x*blocksPerGrid.y*sizeof(int));
  hipMemcpyToSymbol(HIP_SYMBOL(fsum), &ptr, sizeof(ptr));
  funkc<<<blocksPerGrid, threadsPerBlock>>>(M_d, N);
  unsigned int *rez;
  hipMalloc(&rez,1*sizeof(int));
  VecAdd<<<1, blocksPerGrid.x*blocksPerGrid.y>>>(rez,  blocksPerGrid.x*blocksPerGrid.y);  
  hipMemcpy(result, rez, 1*sizeof(int), hipMemcpyDeviceToHost);

  cout<<"rezultat je:"<<result<<endl;

  free(M_h);
  hipFree(M_d);
  hipFree(fsum);

  return 0;
}
