#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>
#include<cmath>

using namespace std;

__global__ void funkc(int *M, int dim)
{
  unsigned int rez;
  __shared__ unsigned int sum;

  int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i < dim && j < dim)
     rez = M[N+i+j]*M[N*i+j];

   __syncthreads();
   atomicAdd(&sum, rez);
   __syncthreads();
}


int main(int argc, char*argv[])
{
  int N(100);

  size_t size = N*N*sizeof(int);

  int *M_h = (int*)malloc(size);

  for(int i(0); i < N*N; i++)
    M_h[i] = i%3;

  int *M_d;
  hipMalloc(&M_d, size);
  hipMemcpy(M_d, M_h, size, hipMemcpyHostToDevice);

  dim3 threadsPerBlock(16,16);
  int blocksPerGrid = ((N / threadsPerBlock.x) + 1, (N/ threadsPerBlock.y) + 1);

  funkc<<<blocksPerGrid, threadsPerBlock>>>(M_d, N);

  free(M_h);
  hipFree(M_d);

  return 0;
}
