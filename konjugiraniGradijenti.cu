#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<fstream>
using namespace std;

int konjugiraniP(double* A, double* b, double* x_0, int dim, double epsilon)
{
	hipblasHandle_t h;
	hipblasCreate(&h);
	double alph(1), bet(-1);
	double tau, result;
	double * d_d, *pom_d, *b_pom_d;
	double *A_d, *b_d, *x_d;
	size_t pitch, dim_d(dim);
	size_t size = dim*sizeof(double);
	int lda_d;

	hipMallocPitch(&A_d, &pitch, size, dim_d);
	hipMalloc(&b_d, size);
	hipMalloc(&x_d, size);
	hipMemcpy2D(A_d, pitch, A, size, size, dim_d, hipMemcpyDefault);
	hipMemcpy(b_d, b, size, hipMemcpyHostToDevice);
	hipMemcpy(x_d, x_0, size, hipMemcpyHostToDevice);
	
	hipMalloc(&d_d, size);
	hipMalloc(&pom_d, size);
	hipMalloc(&b_pom_d,size);
	
	lda_d = pitch/sizeof(double);
	cout<<"lda = "<<lda_d<<endl;

	hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &alph, A_d, lda_d, x_d, 1, &bet, b_d, 1);
	hipblasDcopy(h, dim_d, b_d, 1, d_d, 1);
	hipblasDscal(h, dim_d, &bet, d_d, 1);
	do{
		bet = 0;
		hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &alph, A_d, lda_d, d_d, 1, &bet, pom_d, 1);
		double a, b;
		hipblasDdot(h, dim, b_d, 1, b_d, 1, &a);
		hipblasDdot(h, dim, d_d, 1, pom_d, 1, &b);
		tau = a/b;
		hipblasDaxpy(h, dim, &tau, d_d, 1, x_d, 1);
		hipblasDcopy(h, dim, b_d, 1, b_pom_d, 1);
		hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &tau, A_d, lda_d, d_d, 1, &alph, b_d, 1);
		double beta_k;
		hipblasDdot(h, dim, b_d, 1, b_d, 1, &a);
		hipblasDdot(h, dim, b_pom_d, 1, b_pom_d, 1, &b);
		beta_k = a/b;
		cout<<beta_k<<endl;
		hipblasDcopy(h, dim, b_d, 1, b_pom_d, 1);
		bet = -1;
		hipblasDscal(h, dim, &bet, b_pom_d, 1);
		hipblasDaxpy(h, dim, &beta_k, d_d, 1, b_pom_d, 1);
		hipblasDcopy(h, dim, b_pom_d, 1, d_d, 1);
		hipblasDdot(h, dim, b_d, 1, b_d, 1, &result); 
		cout<<result<<endl;
	}while(result > epsilon);
	
	hipMemcpy(x_0, b_d, size, hipMemcpyDeviceToHost);
	
	cout<<"Zavrsio sam sa cudom"<<endl;
	hipFree(A_d);
	hipFree(x_d);
	hipFree(b_d);
	hipFree(pom_d);
	hipFree(d_d);
	hipFree(b_pom_d);
	hipblasDestroy(h);
	return 1;
}


void procitaj(double *data, int dim, ifstream& file)
{
	for(int i(0); i < dim; i++)
	{
		double dat;
		file>>dat;
		data[i] = dat;
	}
}


int main(int argc, char** argv)
{
	int dim;
	double *A, *b, *x_0;
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	ifstream file( datIme.c_str() );
	
	if( !file.is_open() )
	{
		cerr<<"Greska kod otvaranja datoteke"<<endl;
		exit( -1 );
	}
	
	file>>dim;
	//double *A = (double*)malloc(dim*dim*sizeof(double));
	//double *b = (double*)malloc(dim*sizeof(double));
	//double *x_0 =(double*)malloc(dim*sizeof(double));
	hipHostAlloc(&A, dim*dim*sizeof(double),0);
	hipHostAlloc(&b, dim*sizeof(double),0);
	hipHostAlloc(&x_0, dim*sizeof(double),0);
	double *x_end = (double*)malloc(dim*sizeof(double));
	double epsilon;
	procitaj(A, dim*dim, file);
	procitaj(b, dim, file);
	procitaj(x_0, dim, file);

	cout<<"unesite zadanu tocnost za su sustav :";
	cin>>epsilon;

	if(!konjugiraniP(A, b, x_0, dim, epsilon))
	{
		cout<<"Doslo je do greske kod racuna "<<endl;
		exit ( -1 );
	}
  cout<<"zabrsio sam "<<endl;	
	ofstream rez("rez.txt");
	if(!file.is_open())
	{
		cerr<<"greska kod otvoranja datoteke za rezultat";
		exit(-1);
	}
	for(int i = 0; i < dim; i++)
		rez<<x_0[i]<<endl;
	
	return 0;
}
