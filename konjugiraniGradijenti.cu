#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<fstream>
using namespace std;

int konjugiraniP(double* A, double* b, double* x_0, int dim, double epsilon)
{
	hipblasHandle_t h;
	hipblasCreate(&h);
	double alph(1), bet(-1);
	double tau, beta;
	double * d_d, *pom_d, b_pom_d;
	double *A_d, *b_d, x_d;
	size_t pitch, dim_d(dim);
	int lda_d;
	if(hipMallocPitch(&A_d, &pitch, dim_d*sizeof(double), dim_d) != hipSuccess )
	{
		cerr<<"Greska kod alokacije polja"<<endl;
		exit(-1);
	}

	hipMemcpy2D(A_d,pitch,A,dim*sizeof(double),dim_d*sizeof(double),dim_d,hipMemcpyDefault);
	hipMemcpy(b_d, b, dim_d, hipMemcpyHostToDevice);
	hipMemcpy(x_d, x_0, dim_d, hipMemcpyHostToDevice);
	
	if(hipMalloc(&d_d, dim_d) != hipSuccess || hipMalloc(&pom, dim_d) != hipSuccess \\
		|| hipMalloc(&b_pom_d, dim_d) != hipSuccess)
	{
		cerr<<"Greska kod alokacije za pomocne varijable "<<endl;
		exit(-1);
	}
	
	lda_d = pitch/sizeof(double);

	hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &alph, A_d, lda_d, x_d, 1, &bet, b_d, 1);

	hipblasDestroy(h);
	return 1;
}


void procitaj(double *data, int dim, ifstream& file)
{
	for(int i(0); i < dim; i++)
	{
		double dat;
		file>>dat;
		data[i] = dat;
	}
}


int main(int argc, char** argv)
{
	int dim;
	//double *A, *b, *x_0;
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	ifstream file( datIme.c_str() );
	
	if( !file.is_open() )
	{
		cerr<<"Greska kod otvaranja datoteke"<<endl;
		exit( -1 );
	}
	
	file>>dim;
	double *A = (double*)malloc(dim*dim*sizeof(double));
	double *b = (double*)malloc(dim*sizeof(double));
	double *x_0 =(double*)malloc(dim*sizeof(double));
	double *x_end = (double*)malloc(dim*sizeof(double));
	double epsilon;
	procitaj(A, dim*dim, file);
	procitaj(b, dim, file);
	procitaj(x_0, dim, file);

	cout<<"unesite zadanu tocnost za su sustav :";
	cin>>epsilon;

	if(!konjugiraniP(A, b, x_0, dim, epsilon))
	{
		cout<<"Doslo je do greske kod racuna "<<endl;
		exit ( -1 );
	}
	/*
	ofstream rez("rez.txt");
	if(!file.is_open())
	{
		cerr<<"greska kod otvoranja datoteke za rezultat";
		exit(-1);
	}
	for(int i = 0; i < dim; i++)
		rez<<x_0[i]<<endl;
	*/
	return 0;
}
