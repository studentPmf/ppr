#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<string>
//#include<hip/hip_runtime.h>
//#include<cblas.h>
#include<fstream>
#include<ks.h>
//using namespace std;

/*__global__ 
void VecAdd(double* A, double* B, double* C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( i < N )
		C[i] = A[i] + B[i];
}

int KonjGrad(double *A, double *b, double* x_0, double epsilon, double *x_end, int dim)
{
	double a = cblas_ddot(dim,b,1,x_0,1);
	cout<<a<<endl;
	int k = 0;
	double *r_0 = (double*)malloc(dim*sizeof(double));
	
	return 1;
}
*/


void procitaj(double *data, int dim, ifstream& file)
{
	for(int i(0); i < dim; i++)
	{
		double dat;
		file>>dat;
		data[i] = dat;
	}
}


int main(int argc, char** argv)
{
	int dim;
	//double *A, *b, *x_0;
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	ifstream file( datIme.c_str() );
	
	if( !file.is_open() )
	{
		cerr<<"Greska kod otvaranja datoteke"<<endl;
		exit( -1 );
	}
	
	file>>dim;
	double *A = (double*)malloc(dim*dim*sizeof(double));
	double *b = (double*)malloc(dim*sizeof(double));
	double *x_0 =(double*)malloc(dim*sizeof(double));
	double *x_end = (double*)malloc(dim*sizeof(double));
	double epsilon;
	procitaj(A, dim*dim, file);
	procitaj(b, dim, file);
	procitaj(x_0, dim, file);

	cout<<"unesite zadanu tocnost za su sustav :";
	cin>>epsilon;

	if(!konjugiraniS(A, b, x_0, epsilon, x_end, dim, 0.01))
	{
		cout<<"Doslo je do greske kod racuna "<<endl;
		exit ( -1 );
	}

	return 0;
}
