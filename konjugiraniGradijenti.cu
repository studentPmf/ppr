#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<fstream>
using namespace std;

int konjugiraniP(double* A, double* b, double* x_0, int dim, double epsilon)
{
	hipblasHandle_t h;
	hipblasCreate(&h);
	double alph(1), bet(-1);
	double tau, beta;
	double * d_d, *pom_d, b_pom_d;
	double *A_d, *b_d, *x_d;
	size_t pitch, dim_d(dim);
	int lda_d;
	if(hipMallocPitch(&A_d, &pitch, dim_d*sizeof(double), dim_d) != hipSuccess )
	{
		cerr<<"Greska kod alokacije polja"<<endl;
		exit(-1);
	}

	hipMemcpy2D(A_d,pitch,A,dim*sizeof(double),dim_d*sizeof(double),dim_d,hipMemcpyDefault);
	hipMemcpy(b_d, b, dim_d*sizeof(double), hipMemcpyHostToDevice);
	hipMemcpy(x_d, x_0, dim_d*sizeof(double), hipMemcpyHostToDevice);
	
	if(hipMalloc((void**)&d_d, dim_d*sizeof(double)) != hipSuccess || \\
		 hipMalloc((void**)&pom_d, dim_d*sizeof(double)) != hipSuccess || \\
		 hipMalloc((void**)&b_pom_d, dim_d*sizeof(double)) != hipSuccess)
	{
		cerr<<"Greska kod alokacije za pomocne varijable "<<endl;
		exit(-1);
	}
	
	lda_d = pitch/sizeof(double);

	hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &alph, A_d, lda_d, x_d, 1, &bet, b_d, 1);
	hipblasDcopy(h, dim_d, b_d, 1, d_d, 1);
	hipblasDscal(h, dim_d, &bet, d_d, 1);
	do{
		double result;
		bet = 0;
		hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &alph, A_d, lda_d, d_d, 1, &bet, pom_d, 1);
		double a, b;
		hipblasDdot(h, dim, b_d, 1, b_d, 1, &a);
		hipblasDdot(h, dim, d_d, 1, pom_d, 1, &b);
		tau = a/b;
		hipblasDaxpy(h, dim, &tau, d_d, 1, x_d, 1);
		hipblasDcopy(h, dim, b_d, 1, b_pom_d, 1);
		hipblasDgemv(h, HIPBLAS_OP_N, dim, dim, &tau, A_d, lda_d, d_d, 1, &alph, b_d, 1);
		double beta_k;
		hipblasDdot(h, dim, b_d, 1, b_d, 1, &a);
		hipblasDdot(h, dim, b_pom_d, 1, b_pom_d, 1, &b);
		beta_k = a/b;
		hipblasDcopy(h, dim, b_d, 1, b_pom_d, 1);
		bet = -1;
		hipblasDscal(h, dim, &bet, b_pom_d, 1);
		hipblasDaxpy(h, dim, &beta_k, d_d, 1, b_pom_d, 1);
		hipblasDcopy(h, dim, b_pom_d, 1, d_d, 1);
		hipblasDdot(h, dim, b_d, 1, b_d, 1, &result); 
	}while(result > epsilon)
	
	hipMemcpy(x_0, x_d, dim_d*sizeof(double), hipMemcpyDeviceToHost);

	hipFree(A_d);
	hipFree(x_d);
	hipFree(b_d);
	hipFree(pom_d);
	hipFree(d_d);
	hipFree(b_pom_d);
	hipblasDestroy(h);
	return 1;
}


void procitaj(double *data, int dim, ifstream& file)
{
	for(int i(0); i < dim; i++)
	{
		double dat;
		file>>dat;
		data[i] = dat;
	}
}


int main(int argc, char** argv)
{
	int dim;
	//double *A, *b, *x_0;
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	ifstream file( datIme.c_str() );
	
	if( !file.is_open() )
	{
		cerr<<"Greska kod otvaranja datoteke"<<endl;
		exit( -1 );
	}
	
	file>>dim;
	double *A = (double*)malloc(dim*dim*sizeof(double));
	double *b = (double*)malloc(dim*sizeof(double));
	double *x_0 =(double*)malloc(dim*sizeof(double));
	double *x_end = (double*)malloc(dim*sizeof(double));
	double epsilon;
	procitaj(A, dim*dim, file);
	procitaj(b, dim, file);
	procitaj(x_0, dim, file);

	cout<<"unesite zadanu tocnost za su sustav :";
	cin>>epsilon;

	if(!konjugiraniP(A, b, x_0, dim, epsilon))
	{
		cout<<"Doslo je do greske kod racuna "<<endl;
		exit ( -1 );
	}
	/*
	ofstream rez("rez.txt");
	if(!file.is_open())
	{
		cerr<<"greska kod otvoranja datoteke za rezultat";
		exit(-1);
	}
	for(int i = 0; i < dim; i++)
		rez<<x_0[i]<<endl;
	*/
	return 0;
}
