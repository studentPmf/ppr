#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<cblas.h>
#include<fstream>

using namespace std;

__global__ 
void VecAdd(double* A, double* B, double* C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( i < N )
		C[i] = A[i] + B[i];
}


void procitaj(double *data, int dim, ifstream& file)
{
	for(int i(0); i < dim; i++)
	{
		if(file.eof)
		{
			cerr<<"Greska kod citanja podataka, podatci nisu potpuni"<<endl;
			exit ( -1 );
		}
		double dat;
		file>>dat;
		data[i] = dat;
	}
}


int main(int argc, char** argv)
{
	int dim;
	//double *A, *b, *x_0;
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	ifstream file( datIme.c_str() );
	
	if( !file.is_open() )
	{
		cerr<<"Greska kod otvaranja datoteke"<<endl;
		exit( -1 );
	}
	
	file>>dim;
	double *A = (double*)malloc(dim*dim*sizeof(double));
	double *b = (double*)malloc(dim*sizeof(double));
	double *x_0 =(double*)malloc(dim*sizeof(double));

	procitaj(A, dim*dim, &file);
	procitaj(b, dim, file);
	procitaj(x_0, dim, file);

	return 0;
}
