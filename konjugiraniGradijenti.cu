#include "hip/hip_runtime.h"
#include<iostram>
#include<hip/hip_runtime.h>
#include<cblas.h>

__global__ 
void VecAdd(double* A, double* B, double* C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( i < N )
		C[i] = A[i] + B[i];
}


int main(int argc, char** argv)
{
	return 0;
}
