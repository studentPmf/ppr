#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<cblas.h>
#include<fstream>

using namespace std;

__global__ 
void VecAdd(double* A, double* B, double* C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( i < N )
		C[i] = A[i] + B[i];
}


void procitaj(double *data, int dim, ifstream file)
{
	for(int i(0); i < dim; i++)
	{
		if(file.eof)
		{
			cerr<<"Greska kod citanja podataka, podatci nisu potpuni"<<endl;
			exit ( -1 );
		}
		double dat;
		file>>dat;
		data[i] = data;
	}
}


int main(int argc, char** argv)
{
	int dim;
	double * A, *b, *x_0;
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	ifstream file( datIme.c_str() );
	
	if( !file.is_open() )
	{
		cerr<<"Greska kod otvaranja datoteke"<<endl;
		exit( -1 );
	}
	
	file>>dim;
	*A = new int[dim*dim];
	*b = new int[dim];
	*x_0 = new int[dim];

	procitaj(A, dim*dim, file);
	procitaj(b, dim, file);
	procitaj(x_0, dim, file);

	return 0;
}
