#include<iostream>
#include<string>
#include<hip/hip_runtime.h>
#include<cblas.h>
#include<fstream>

using namespace std;

__global__ 
void VecAdd(double* A, double* B, double* C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( i < N )
		C[i] = A[i] + B[i];
}


int main(int argc, char** argv)
{
	std::string datIme;
	std::cout<<"Unesite ime tekstualne datoteke u kojoj se nalazi zadani sustav: ";
	std::cin>>datIme;
	std::cout<<std::endl;
	std::cout<<"Unijeli ste ime "<<datIme;
	std::cout<<"i rezltat ce biti spremnjen u datoteku rez.txt"<<std::endl;

	fstream file(datIme.c_str());

	return 0;
}
