#include<iostream>
#include<hip/hip_runtime.h>

__global__ void VecAdd(double *A, double *B, double *C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( N > i )
		C[i] = A[i] + B[i];
}


int main(int argc, char *argv[])
{
	int N = 100;
	
	size_t size = N * sizeof(double);

	double *h_A = (double*)malloc(size);
	double *h_B = (double*)malloc(size);

	for(int i(0); i < 100; i++)
		h_A[i] = h_B[i] = (double)( (i + 2)/(i + 1) );
	
	double *d_A, *d_B, *d_C;
	hipMalloc( &d_A, size);
	hipMalloc( &d_B, size);
	hipMalloc( &d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
	int threadsPerBlock = 20;
  int blocksPerGrid = 5;
   
	VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

	double *h_C = (double*)malloc(size);
	hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

	for(int i = 0; i < N; i++)
		std::cout<<h_C[i]<<" ";
	std::cout<<std::endl;

	hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);
		
	
	free(h_A);
	free(h_B);

	return 0;
}
