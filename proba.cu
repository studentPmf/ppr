#include "hip/hip_runtime.h"
#include<iostream>
#include<hip/hip_runtime.h>

__global__ void VecAdd(double *A, double *B, double *C, int N)
{
	int i = blockDim.x * blockIdx.x + threadIdx.x;

	if( N > i )
		C[i] = A[i] + B[i];
}


int main(int argc, char *argv[])
{
	int N(100);
	size_t size = N * sizeof(double);

	double *h_A = (double*)malloc(size);
	double *h_B = (double*)malloc(size);

	for(int i(0); i < 100; i++)
		h_A[i] = h_B[i] = (double)( (i + 2)/(i + 1) );
	
	double *d_A, *d_B, *d_C;
	hipMalloc(&amp; d_A, size);
	hipMalloc(&amp; d_B, size);
	hipMalloc(&amp; d_C, size);

	hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
	
	 int threadsPerBlock = 256;
   int blocksPerGrid = (N + threadsPerBlock – 1) / threadsPerBlock;
   
	 VecAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);

	 hipFree(d_A);
   hipFree(d_B);
   hipFree(d_C);

	 free(h_A);
	 free(h_B);

	return 0;
}
