#include "hip/hip_runtime.h"
/**
Napisati CUDA program koji trazi maksimalni nezavisni skup vrhova
u grafu korištenjem paralelnog algoritma koji koristi slučajne brojeve.
*/
#include<iostream>
#include<stdio.h>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hipblas.h>
#include<fstream>
#include<time.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <time.h>
using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

/**
  * Host funkcija koja provjerava koliko je ostalo 
  * neodabranih vrhova
  */
bool findZeros(int* polje, int n)
{
  for(int i = 0; i < n; i++)
    if(polje[i] == 0)
      return true;

  return false;
}

/**
  * Umnozak pseudo i vrijeme
  */
__global__ void bestRand(float *devData, int* n)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < *n)
    devData[index] = devData[index]*((int)clock()%10);
}

/**
  * Curand host funkcija. Generira pseudo-slucajne brojeve <0,1>
  * uniformna razdioba
  */

int create_pseud_numbers(float *hostData, float *devData, int numElements)
{
  size_t n = numElements;
  hiprandGenerator_t gen;
  //int *nn;
  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                1234ULL));

  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

  /*CUDA_CALL(hipMalloc((void**)&nn, sizeof(int)));
  CUDA_CALL(hipMemcpy(nn, &n, sizeof(int),
        hipMemcpyHostToDevice));
  bestRand<<<n/128 + 1,128>>>(devData,nn);
  */
  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
        hipMemcpyDeviceToHost));

  /* Cleanup */
  CURAND_CALL(hiprandDestroyGenerator(gen));

  return EXIT_SUCCESS;
}

/**
  * Device funkcija. Algoritam za pronalazenje maksimalnog nezavisnog skupa vrhova.
  * Ulazni parametri : polje veza, polje pokazivaca na veze za svaki vrh po jedan pointer na polje veze,
  *                    polje izbaceni, svaki thread zapise tko je izbacen sa -1 a ako je on trazeni postavi 1
  */

__global__ void algoritam(int* veze, int* ptr, int* izbaceni, float *devData, int* veze_size, int* ptr_size)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int c =(int)clock()%10;
  index = (index + c)%ptr_size;
  //izbaceni[index] = index; //provjera indekasa

  // Ako ti je index u rangu i ako nisi vec izbacen
  if(index < *ptr_size - 1 && izbaceni[index] != -1)
  {
    int provjera = 1;
    int start = ptr[index]; //pocetak u vezama
    int end = ptr[index + 1]; // kraj u vezama
    for(int i = start; i < end; i++)
    {
      // Ako je netko dobio vecu tezinu i ako taj nije izbacen kao mogucnost
      if(devData[index] >= devData[veze[i] - 1] && izbaceni[veze[i] - 1] != -1)
        provjera = 0;
    }

    // Ako je prosao provjeru
    if(provjera)
    {
       izbaceni[index] = 1; // postavi da je index dobar
       for(int i = start; i< end; i++)
         izbaceni[veze[i] - 1 ] = -1; // sve susjede izbaci kao mogucnost
    }
  }
}


int main(int argc, const char* argv[])
{
  // Provjera da li su dobri ulazni parametri
  if( argc != 2)
  {
    cerr<<"Krivi ulazni parametri"<<endl;
    return EXIT_FAILURE;
  }

  int numElements; // broj vrhova
  vector<int> indElements; // vektor susjedstva ( veze )
  vector<int> ptrVector; // pointeri vrhova u vektoru susjedstva
  ifstream myFile (argv[1]);
  
  // Ako je file dobar prepisi ga u vektore
  if(myFile.is_open())
  {
    int cnt = 0;
    myFile >> numElements;
    ptrVector.push_back(cnt);
    while(myFile.good())
    {
      int v;
      myFile >> v;
      if(!myFile.good()) break;
      if(v == 0)
        ptrVector.push_back(cnt);
      else
      {
        indElements.push_back(v);
        cnt++;
      }
    }
  }
  else
  {
    cerr<<"Pogresno ime datoteke"<<endl;
    return EXIT_FAILURE;
  }

  /* Provjera da li je sve procitano korektno*/
  /*cout<<numElements<<endl;
  for(int i(0); i < ptrVector.size(); i++)
    cout<<ptrVector[i]<<" ";
  cout<<endl;

  for(int i(0); i < indElements.size(); i++)
    cout<<indElements[i]<<" ";
  cout<<endl;
  */
  
  /* Priprema za device*/
  /*****************************************************************/

  int* HindElements = &indElements[0]; // iz vektora u polje
  int* HptrVector = &ptrVector[0];     // iz vektora u polje
  int Hizbaceni[numElements];
  // Inicijalno sve na 0 jer su svi vrhovi raspolozivi za koristenje
  for(int i(0); i < numElements; i++)
    Hizbaceni[i] = 0;

  float * hostData, *devData; // polja za pseudo-slucajne brojeve
  
  // alokacija za generator pseudo brojeva
  hostData = (float *)calloc(numElements, sizeof(float));
  CUDA_CALL(hipMalloc((void **)&devData, numElements*sizeof(float)));
  
  create_pseud_numbers(hostData, devData, numElements);
  
  /* Prikaz rezultata */
  /*
  for( int i = 0; i < numElements; i++) {
    printf("%1.4f ", hostData[i]);
  }
  cout<<endl;
  */
  // Alokacija memorija za glavni program (algoritam)
  int Hveze_size = indElements.size(), Hptr_size = ptrVector.size(); // pomocne varijable  
  int *Dveze_size, *Dptr_size;
  int *DindElements, *DptrVector, *Dizbaceni;
  int izbaceni[numElements];

  CUDA_CALL(hipMalloc((void **)&DindElements, indElements.size()*sizeof(int)));
  CUDA_CALL(hipMalloc((void **)&DptrVector, ptrVector.size()*sizeof(int)));
  CUDA_CALL(hipMalloc((void **)&Dizbaceni, numElements*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Dveze_size, sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Dptr_size, sizeof(int)));


  CUDA_CALL(hipMemcpy(DindElements, HindElements, indElements.size() * sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(DptrVector, HptrVector, ptrVector.size() * sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Dizbaceni, &Hizbaceni, numElements * sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Dveze_size, &Hveze_size, sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Dptr_size, &Hptr_size, sizeof(int),
        hipMemcpyHostToDevice));
 

  // CUDA grid
  //dim3 threadsPerBlock(16, 16);
  //dim3 numBlocks(numElements / threadsPerBlock.x, numElements / threadsPerBlock.y);

  // Algoritam
  do{
    
    algoritam<<<numElements/128 + 1 ,128>>>(DindElements, DptrVector, Dizbaceni, devData, Dveze_size, Dptr_size);
    CUDA_CALL(hipMemcpy(izbaceni, Dizbaceni, numElements * sizeof(int), hipMemcpyDeviceToHost));

  }while(findZeros(izbaceni, numElements));

  // ispisi matrice odabranih i izbacenih vrhova 1 -> odabrani, -1 -> izbaceni
  for( int k = 0; k < numElements; k++)
    cout<<k+1<<" : "<<izbaceni[k]<<endl;

  // Oslobadanje memorije na hostu i divace-u 
  free(hostData);
  CUDA_CALL(hipFree(devData));
  CUDA_CALL(hipFree(DindElements));
  CUDA_CALL(hipFree(DptrVector));
  CUDA_CALL(hipFree(Dizbaceni));
  CUDA_CALL(hipFree(Dveze_size));
  CUDA_CALL(hipFree(Dptr_size));

  return 0;
}
