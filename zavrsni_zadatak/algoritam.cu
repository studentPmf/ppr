#include<iostream>
#include<stdio.h>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hipblas.h>
#include<fstream>
#include<time.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

void create_pseud_numbers(float *hostData, int numElements)
{
  size_t n = 5;//numElements;
  hiprandGenerator_t gen;
  float *devData;

  /* Allocate n floats on host */
  hostData = (float *)calloc(n, sizeof(float));

  /* Allocate n floats on device */
  CUDA_CALL(hipMalloc((void **)&devData, n*sizeof(float)));

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                1234ULL));

  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
        hipMemcpyDeviceToHost));

  /* Cleanup */
  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipFree(devData));
}


int main(int argc, const char* argv[])
{
  // Provjera da li su dobri ulazni parametri
  if( argc != 2)
  {
    cerr<<"Krivi ulazni parametri"<<endl;
    return EXIT_FAILURE;
  }

  /* Citanje iz datoteke
     Sve se sprema u vektor
     Ime datoteke se cita s kom. linije
  */
  int numElements;
  thrust::host_vector<int> indElements; // vektor veza za sve vrhove, format v1v20v1v3v40...
  thrust::host_vector<int> ptrVector;
  ifstream myFile (argv[1]);
  
  if(myFile.is_open())
  {
    int cnt = 0;
    myFile >> numElements;
    ptrVector.push_back(cnt);
    while(myFile.good())
    {
      int v;
      myFile >> v;
      if(v == 0)
        ptrVector.push_back(cnt);
      else
      {
        indElements.push_back(v);
        cnt++;
      }
    }
  }
  else
  {
    cerr<<"Pogresno ime datoteke"<<endl;
    return EXIT_FAILURE;
  }
  
  /*for(int i(0); i < indElements.size(); i++)
    cout<<indElements[i];
  cout<<endl;*/
  //********************************************//
  
  thrust::device_vector<int> DindElements = indElements; // vektor elemenata
  thrust::device_vector<int> DptrVector = ptrVector;     // vektor pointera na pocetak za svaki vrh
  
  float * hostData;
  
  //for(int j = 0; j < 5; j++)
  //{
    create_pseud_numbers(hostData, numElements);
    /* Show result */
    for( int i = 0; i < numElements; i++) {
      printf("%1.4f ", hostData[i]);
    }
    cout<<endl;
    free(hostData);
  //}

}
