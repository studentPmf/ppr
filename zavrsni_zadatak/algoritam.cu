#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<fstream>
#include<time.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

__global__ void postavi_tezine(double tezine)
{
  
}


int main(int argc, const char* argv[])
{
  // Provjera da li su dobri ulazni parametri
  if( argc != 2)
  {
    cerr<<"Krivi ulazni parametri"<<endl;
    return EXIT_FAILURE;
  }

  /* Citanje iz datoteke
     Sve se sprema u host_vektor
     Ime datoteke se cita s kom. linije
  */
  thrust::host_vector<thrust::host_vector<int> > indElements; 
  string fileName = argv[1];
  ifstream myFile (string);
  
  if(myFile.is_open())
  {
    int numElements;
    myFile >> numElements;
    while(myFile.good())
    {
      int v;
      thrust::host_vector<int> pom;
      myFile >> v;
      if( v == 0)
      {
        indElements.push_back(pom);
        pom.clear();
      }
      else
        pom.push_back(v);
    }
  }
  else
  {
    cerr<<"Pogresno ime datoteke"<<endl;
    return EXIT_FAILURE;
  }

  //********************************************//


 return EXIT_SUCCESS;
}
