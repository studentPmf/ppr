#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hipblas.h>
#include<fstream>
#include<time.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

int create_pseud_numbers(float *hostData, float *devData, int numElements)
{
  size_t n = numElements;
  hiprandGenerator_t gen;
  //float *devData;

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                1234ULL));

  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
        hipMemcpyDeviceToHost));

  /* Cleanup */
  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipFree(devData));

  return EXIT_SUCCESS;
}

__global__ void algoritam(int* veze, int* ptr, int* izbaceni, float *devData, int* veze_size, int* ptr_size)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < *ptr_size && izbaceni[index] != -1)
  {
    int start, end;
    int provjera = 1;
    start = ptr[index];
    if( index + 1 >= *ptr_size)
      end = *veze_size;
    else
      end = ptr[index + 1];
    for(int i = start; i < end; i++)
    {
      if(devData[index] > devData[veze[i]])
        provjera = 0;
    }

    if(provjera)
    {
       izbaceni[index] = 1;
       for(int i = start; i< end; i++)
         izbaceni[i] = -1;
    }
  }
}

int main(int argc, const char* argv[])
{
  // Provjera da li su dobri ulazni parametri
  if( argc != 2)
  {
    cerr<<"Krivi ulazni parametri"<<endl;
    return EXIT_FAILURE;
  }

  /* Citanje iz datoteke
     Sve se sprema u vektor
     Ime datoteke se cita s kom. linije
  */
  int numElements;
  thrust::host_vector<int> indElements; // vektor veza za sve vrhove, format v1v20v1v3v40...
  thrust::host_vector<int> ptrVector;
  ifstream myFile (argv[1]);
  
  if(myFile.is_open())
  {
    int cnt = 0;
    myFile >> numElements;
    ptrVector.push_back(cnt);
    while(myFile.good())
    {
      int v;
      myFile >> v;
      if(v == 0)
        ptrVector.push_back(cnt);
      else
      {
        indElements.push_back(v);
        cnt++;
      }
    }
  }
  else
  {
    cerr<<"Pogresno ime datoteke"<<endl;
    return EXIT_FAILURE;
  }
  
  for(int i(0); i < indElements.size(); i++)
    cout<<indElements[i];
  cout<<endl;
  //********************************************//
  
  int* HindElements = &indElements[0]; // vektor elemenata
  int* HptrVector = &ptrVector[0];     // vektor pointera na pocetak za svaki vrh
  int Hizbaceni[numElements];
  memset(&Hizbaceni,0,numElements);
  float * hostData, *devData;
  int Hveze_size = indElements.size(), Hptr_size = ptrVector.size();
  int *Dveze_size, *Dptr_size;
    /* Allocate n floats on host */
  hostData = (float *)calloc(numElements, sizeof(float));
    /* Allocate n floats on device */
  CUDA_CALL(hipMalloc((void **)&devData, numElements*sizeof(float)));

  create_pseud_numbers(hostData, devData, numElements);
  
  /* Show result */
  for( int i = 0; i < numElements; i++) {
    printf("%1.4f ", hostData[i]);
  }
  cout<<endl;
   
  /*
  int *DindElements,*DptrVector, *Dizbaceni;
  CUDA_CALL(hipMalloc((void **)&DindElements, indElements.size()*sizeof(int)));
  CUDA_CALL(hipMalloc((void **)&DptrVector, ptrVector.size()*sizeof(int)));
  CUDA_CALL(hipMalloc((void **)&Dizbaceni, numElements*sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Dveze_size, sizeof(int)));
  CUDA_CALL(hipMalloc((void**)&Dptr_size, sizeof(int)));


  CUDA_CALL(hipMemcpy(DindElements, HindElements, indElements.size() * sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(DptrVector, HptrVector, ptrVector.size() * sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Dizbaceni, &Hizbaceni, numElements * sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Dveze_size, &Hveze_size, sizeof(int),
        hipMemcpyHostToDevice));
  CUDA_CALL(hipMemcpy(Dptr_size, &Hptr_size, sizeof(int),
        hipMemcpyHostToDevice));

  //algoritam<<<1,numElements>>>(DindElements, DptrVector, Dizbaceni, devData, Dveze_size, Dptr_size);
  int *izbaceni;
  CUDA_CALL(hipMemcpy(izbaceni, Dizbaceni, numElements * sizeof(int),
        hipMemcpyDeviceToHost));
  */
  /*for( int k = 0; k < numElements; k++)
    cout<<izbaceni[k];
  cout<<endl;*/
  free(hostData);
  
}
