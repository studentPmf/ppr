#include "hip/hip_runtime.h"
#include<iostream>
#include<stdio.h>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hip/hip_runtime.h>
#include<hiprand.h>
#include<hipblas.h>
#include<fstream>
#include<time.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

#define CUDA_CALL(x) do { if((x)!=hipSuccess) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)
#define CURAND_CALL(x) do { if((x)!=HIPRAND_STATUS_SUCCESS) { \
    printf("Error at %s:%d\n",__FILE__,__LINE__);\
    return EXIT_FAILURE;}} while(0)

int create_pseud_numbers(float *hostData, float *devData, int numElements)
{
  size_t n = numElements;
  hiprandGenerator_t gen;
  //float *devData;

  /* Create pseudo-random number generator */
  CURAND_CALL(hiprandCreateGenerator(&gen, 
                HIPRAND_RNG_PSEUDO_DEFAULT));

  /* Set seed */
  CURAND_CALL(hiprandSetPseudoRandomGeneratorSeed(gen, 
                1234ULL));

  /* Generate n floats on device */
  CURAND_CALL(hiprandGenerateUniform(gen, devData, n));

  /* Copy device memory to host */
  CUDA_CALL(hipMemcpy(hostData, devData, n * sizeof(float),
        hipMemcpyDeviceToHost));

  /* Cleanup */
  CURAND_CALL(hiprandDestroyGenerator(gen));
  CUDA_CALL(hipFree(devData));

  return EXIT_SUCCESS;
}

__global__ void algoritam(thrust::device_vector<int> veze, thrust::device_vector<int> ptr, thrust::device_vector<int>& izbaceni, float *devData)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if(index < ptr.size() && izbaceni[index] != -1)
  {
    int start, end;
    int provjera = 1;
    start = ptr[index];
    if( index + 1 >= ptr.size())
      end = veze.size();
    else
      end = ptr[index + 1];
    for(int i = start; i < end; i++)
    {
      if(devData[index] > devData[veze[i]])
        provjera = 0;
    }

    if(provjera)
    {
       izbaceni[index] = 1;
       for(int i = start; i< end; i++)
         izbaceni[i] = -1;
    }
  }
}

int main(int argc, const char* argv[])
{
  // Provjera da li su dobri ulazni parametri
  if( argc != 2)
  {
    cerr<<"Krivi ulazni parametri"<<endl;
    return EXIT_FAILURE;
  }

  /* Citanje iz datoteke
     Sve se sprema u vektor
     Ime datoteke se cita s kom. linije
  */
  int numElements;
  thrust::host_vector<int> indElements; // vektor veza za sve vrhove, format v1v20v1v3v40...
  thrust::host_vector<int> ptrVector;
  ifstream myFile (argv[1]);
  
  if(myFile.is_open())
  {
    int cnt = 0;
    myFile >> numElements;
    ptrVector.push_back(cnt);
    while(myFile.good())
    {
      int v;
      myFile >> v;
      if(v == 0)
        ptrVector.push_back(cnt);
      else
      {
        indElements.push_back(v);
        cnt++;
      }
    }
  }
  else
  {
    cerr<<"Pogresno ime datoteke"<<endl;
    return EXIT_FAILURE;
  }
  
  /*for(int i(0); i < indElements.size(); i++)
    cout<<indElements[i];
  cout<<endl;*/
  //********************************************//
  
  thrust::device_vector<int> DindElements = indElements; // vektor elemenata
  thrust::device_vector<int> DptrVector = ptrVector;     // vektor pointera na pocetak za svaki vrh
  thrust::device_vector<int> izbaceni;
  izbaceni.assign(0,numElements);
  float * hostData, *devData;
    /* Allocate n floats on host */
  hostData = (float *)calloc(numElements, sizeof(float));
    /* Allocate n floats on device */
  CUDA_CALL(hipMalloc((void **)&devData, numElements*sizeof(float)));

  create_pseud_numbers(hostData, devData, numElements);
  
  /* Show result */
  /*for( int i = 0; i < numElements; i++) {
    printf("%1.4f ", hostData[i]);
  }
  cout<<endl;*/

  algoritam<<<1,numElements>>>(indElements, ptrVector, izbaceni, devData);
  free(hostData);
  
}
