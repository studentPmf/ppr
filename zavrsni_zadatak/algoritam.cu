#include "hip/hip_runtime.h"
#include<iostream>
#include<cstdlib>
#include<string>
#include<hip/hip_runtime.h>
#include<hipblas.h>
#include<fstream>
#include<time.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
using namespace std;

__global__ void postavi_tezine(double tezine)
{
  
}


int main(int argc, const char* argv[])
{
  // Provjera da li su dobri ulazni parametri
  if( argc != 2)
  {
    cerr<<"Krivi ulazni parametri"<<endl;
    return EXIT_FAILURE;
  }

  /* Citanje iz datoteke
     Sve se sprema u vektor
     Ime datoteke se cita s kom. linije
  */
  thrust::host_vector<int> indElements; // vektor veza za sve vrhove, format v1v20v1v3v40...
  ifstream myFile (argv[1]);
  
  if(myFile.is_open())
  {
    int numElements;
    myFile >> numElements;
    while(myFile.good())
    {
      int v;
      myFile >> v;
      indElements.push_back(v);
    }
  }
  else
  {
    cerr<<"Pogresno ime datoteke"<<endl;
    return EXIT_FAILURE;
  }

  //********************************************//



}
